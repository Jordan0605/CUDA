/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

int nsteps, tpoints;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n",
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

__global__ void exe(float* finalval, int tpoints, int nsteps){
  float values, newval, oldval;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //init
  float x = (float)(i - 1) / (tpoints - 1);
  values = sin(2.0 * PI * x);
  oldval = values;
  //update
  if(i == 0 || i == tpoints) values = 0;
  else{
    for(int i=1 ; i<=nsteps ; i++){
      //tau = 0.3 , sqtau = 0.09 , new = 2 * values - oldval + 0.09 * (-2) * values
      //new = 1.82 * values - oldval
      newval = 1.82 * values - oldval;
      oldval = values;
      values = newval;
    }
  }
  finalval[i] = values;
}

void printfinal(float* values)
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

int main(int argc, char *argv[])
{
  float finalval[MAXPOINTS + 2];
  float* final_D;
  int size = (MAXPOINTS + 2) * sizeof(float);
  hipMalloc((void**)&final_D, size);

	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
  int threadPerBlock = 1024;
  int numBlock = tpoints / threadPerBlock + 1;
	//update();
  exe<<<numBlock,threadPerBlock>>>(final_D, tpoints, nsteps);

  hipMemcpy(finalval, final_D, size, hipMemcpyDeviceToHost);
  hipFree(final_D);

	printf("Printing final results...\n");
	printfinal(finalval);
	printf("\nDone.\n\n");

	return 0;
}
